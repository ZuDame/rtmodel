
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

struct Voxel {
    short int sd;
    short int w;
};

__global__ void incr_tsdf(Voxel *vox)
{
   for (int z = 0; z < 512; z++) {
      int idx = z*512*512 + blockIdx.x*512 + threadIdx.x;
      vox[idx].sd += threadIdx.x;
      vox[idx].w += blockIdx.x;
   }
}

int main(void) {
    Voxel *vox_gpu;
    Voxel *vox_cpu;

    vox_cpu = (Voxel *) malloc(512*512*512*4);
    hipMalloc((void **) &vox_gpu, 512*512*512*4);
    dim3 dimBlock(512,1,1);
    dim3 dimGrid(512,1,1);

    int N = 10;

    hipEvent_t e_start, e_stop;
    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);
    hipEventRecord(e_start);
    for (int i = 0; i < N; i++) {      
	incr_tsdf<<<dimGrid, dimBlock>>>(vox_gpu);
    }
    hipEventRecord(e_stop);
    hipEventSynchronize(e_stop);

    float ms;
    hipEventElapsedTime(&ms, e_start, e_stop);

    hipMemcpy(vox_cpu, vox_gpu, 512*512*512*4, hipMemcpyDeviceToHost);
    for (int i = 0; i < 20; i++) {
    	printf("[%03d] %d %d\n", i, (int) vox_cpu[i].sd, (int) vox_cpu[i].w);
    }

    printf("%d in %.1f (avg %.1f)\n", N, ms, ms/N);

    hipFree(vox_gpu);
    free(vox_cpu);
    return 0;
}
